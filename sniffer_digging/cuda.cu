// !nvcc -O3 -arch=sm_75 cuda.cu -o cuda

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <inttypes.h>

#define MASK_64 0xFFFFFFFFFFFFFFFFULL
#define SILVER_RATIO_64 0x6a09e667f3bcc909ULL
#define SUBTRACT_CONST 0x61C8864680B583EBULL
#define SNFFER_DIG_MD5_0 0x20bd0d779a2856fcULL
#define SNFFER_DIG_MD5_1 0x491767dc04f0109aULL
#define STAFFORD_MIX_1 0xbf58476d1ce4e5b9ULL
#define STAFFORD_MIX_2 0x94d049bb133111ebULL
#define MIN_SEQUENCE 39
#define TOTAL_WEIGHT 2

struct Xoroshiro128Plus {
    uint64_t l, h;

    __device__ __forceinline__ Xoroshiro128Plus(uint64_t seed) {
        uint64_t unmixed_lo = seed ^ SILVER_RATIO_64;
        uint64_t unmixed_hi = (unmixed_lo - SUBTRACT_CONST) & MASK_64;

        l = mix_stafford(unmixed_lo ^ SNFFER_DIG_MD5_0);
        h = mix_stafford(unmixed_hi ^ SNFFER_DIG_MD5_1);
    }

    __device__ __forceinline__ uint64_t mix_stafford(uint64_t x) {
        x = (x ^ (x >> 30)) * STAFFORD_MIX_1;
        x = (x ^ (x >> 27)) * STAFFORD_MIX_2;
        x ^= x >> 31;
        return x & MASK_64;
    }

    __device__ __forceinline__ uint64_t rotl(uint64_t x, int k) {
        return ((x << k) | (x >> (64 - k))) & MASK_64;
    }

    __device__ __forceinline__ uint64_t next_long() {
        uint64_t s0 = l;
        uint64_t s1 = h;
        uint64_t result = ((rotl((s0 + s1), 17) + s0) & MASK_64);
        s1 ^= s0;
        l = (rotl(s0, 49) ^ s1 ^ ((s1 << 21) & MASK_64)) & MASK_64;
        h = rotl(s1, 28) & MASK_64;
        return result;
    }

    __device__ __forceinline__ int next_int_bound(int bound) {
        uint64_t l, m, low;
        do {
            l = next_long() & 0xFFFFFFFFULL;
            m = l * bound;
            low = m & 0xFFFFFFFFULL;
        } while (low < bound && low < ((-bound) % bound));
        return (int)((m >> 32) & 0xFFFFFFFFULL);
    }

    __device__ __forceinline__ int roll_sniffer() {
        return next_int_bound(TOTAL_WEIGHT);
    }
};

__global__ void search(uint64_t start, uint64_t end) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    for (uint64_t seed = start + idx; seed < end; seed += stride) {
        Xoroshiro128Plus rng(seed);

        int streak = 0;
        while (true) {
            int drop = rng.roll_sniffer();
            if (drop == 0) { // back to back torchflower if 0. back to back pitchre pod if 1.
                streak++;
            } else {
                break;
            }
        }

        if (streak >= MIN_SEQUENCE) {
            printf("Seed %" PRIu64 " with %d\n", seed, streak);
        }
    }
}

int main() {
    uint64_t start = 0ULL;
    uint64_t end   = 1000000000000ULL;

    int threads = 256;
    int blocks  = 256;

    search<<<blocks, threads>>>(start, end);
    hipDeviceSynchronize();

    return 0;
}
