// !nvcc -O3 -arch=sm_75 backappleoak.cu -o backappleoak
// Finding back-to-back apples from oak leaves

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <inttypes.h>

#define MASK_64          0xFFFFFFFFFFFFFFFFULL
#define SILVER_RATIO_64  0x6A09E667F3BCC909ULL
#define SUBTRACT_CONST   0x61C8864680B583EBULL
#define OAK_MD5_0        0xEF6489BEC2529E35ULL
#define OAK_MD5_1        0x1F1AB2C703AA2B5DULL
#define STAFFORD_MIX_1   0xBF58476D1CE4E5B9ULL
#define STAFFORD_MIX_2   0x94D049BB133111EBULL
#define MIN_STREAK       5       // min back-to-back it outputs

struct Xoroshiro128Plus {
    uint64_t l, h;

    __device__ __forceinline__ Xoroshiro128Plus(uint64_t seed) {
        uint64_t unmixed_lo = seed ^ SILVER_RATIO_64;
        uint64_t unmixed_hi = (unmixed_lo - SUBTRACT_CONST) & MASK_64;

        l = mix_stafford(unmixed_lo ^ OAK_MD5_0);
        h = mix_stafford(unmixed_hi ^ OAK_MD5_1);
    }

    __device__ __forceinline__ uint64_t mix_stafford(uint64_t x) {
        x = (x ^ (x >> 30)) * STAFFORD_MIX_1;
        x = (x ^ (x >> 27)) * STAFFORD_MIX_2;
        x ^= x >> 31;
        return x & MASK_64;
    }

    __device__ __forceinline__ uint64_t next_long() { // turn l and h into 64 bit #
        uint64_t s0 = l;
        uint64_t s1 = h;
        uint64_t result = ((rotl((s0 + s1), 17) + s0) & MASK_64);
        s1 ^= s0;
        l = (rotl(s0, 49) ^ s1 ^ ((s1 << 21) & MASK_64)) & MASK_64;
        h = rotl(s1, 28) & MASK_64;
        return result;
    }

    __device__ __forceinline__ uint32_t next_int32() {
        return next_long() & 0xFFFFFFFFULL;
    }

    __device__ __forceinline__ float next_float24() {
        return (next_long() >> 40) / float(1 << 24);
    }

    __device__ __forceinline__ bool next_has_apple() {
        const float stick_chance[5]   = {0.02f, 0.022222223f, 0.025f, 0.033333335f, 0.1f};
        const float apple_chance[5]   = {0.005f, 0.0055555557f, 0.00625f, 0.008333334f, 0.025f};

        int fortune = 0; // change if u want but no fortune is better

        next_float24();  

        if (next_float24() < stick_chance[fortune]) {
            next_int32(); 
        }
        // apples. not my favorite fruit
        return next_float24() < apple_chance[fortune];
    }

    __device__ __forceinline__ uint64_t rotl(uint64_t x, int k) {
        return ((x << k) | (x >> (64 - k))) & MASK_64;
    }
};

__global__ void search(uint64_t start, uint64_t end) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    for (uint64_t seed = start + idx; seed < end; seed += stride) {
        Xoroshiro128Plus rng(seed);

        int streak = 0;
        while (rng.next_has_apple()) {
            streak++;
        }

        if (streak >= MIN_STREAK) {
            printf("Seed %" PRIu64 " with %d\n", seed, streak);
        }
    }
}

int main() {
    uint64_t start = 0ULL;
    uint64_t end   = 1000000000000ULL;

    int threads = 256;
    int blocks  = 256;

    search<<<blocks, threads>>>(start, end);
    hipDeviceSynchronize();

    return 0;
}