// !nvcc -O3 -arch=sm_75 unluckyblaze.cu -o unluckyblaze

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <inttypes.h>

#define MASK_64         0xFFFFFFFFFFFFFFFFULL
#define SILVER_RATIO_64 0x6a09e667f3bcc909ULL
#define SUBTRACT_CONST  0x61C8864680B583EBULL
#define BLAZE_MD5_0     0xa9ec152f9c889472ULL
#define BLAZE_MD5_1     0xcb9b0580c2b91a9eULL
#define STAFFORD_MIX_1  0xbf58476d1ce4e5b9ULL
#define STAFFORD_MIX_2  0x94d049bb133111ebULL
#define MIN_SEQUENCE    35       // minimum streak required

struct Xoroshiro128Plus {
    uint64_t l, h;

    __device__ __forceinline__ Xoroshiro128Plus(uint64_t seed) {
        uint64_t unmixed_lo = seed ^ SILVER_RATIO_64;
        uint64_t unmixed_hi = (unmixed_lo - SUBTRACT_CONST) & MASK_64;

        l = mix_stafford(unmixed_lo ^ BLAZE_MD5_0);
        h = mix_stafford(unmixed_hi ^ BLAZE_MD5_1);
    }

    __device__ __forceinline__ uint64_t mix_stafford(uint64_t x) {
        x = (x ^ (x >> 30)) * STAFFORD_MIX_1;
        x = (x ^ (x >> 27)) * STAFFORD_MIX_2;
        x ^= x >> 31;
        return x & MASK_64;
    }

    __device__ __forceinline__ uint64_t rotl(uint64_t x, int k) {
        return ((x << k) | (x >> (64 - k))) & MASK_64;
    }

    __device__ __forceinline__ uint64_t next_long() {
        uint64_t s0 = l;
        uint64_t s1 = h;
        uint64_t result = ((rotl((s0 + s1), 17) + s0) & MASK_64);
        s1 ^= s0;
        l = (rotl(s0, 49) ^ s1 ^ ((s1 << 21) & MASK_64)) & MASK_64;
        h = rotl(s1, 28) & MASK_64;
        return result;
    }

    __device__ __forceinline__ uint32_t next_int32() {
        return next_long() & 0xFFFFFFFFULL;
    }

    __device__ __forceinline__ bool next_blaze_rod() {
        return ((next_int32() >> 31) % 2) == 1;
    }
};

__global__ void search(uint64_t start, uint64_t end) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    for (uint64_t seed = start + idx; seed < end; seed += stride) {
        Xoroshiro128Plus rng(seed);

        int streak = 0;

        while (!rng.next_blaze_rod()) {
            streak++;
        }

        if (streak >= MIN_SEQUENCE) {
            printf("Seed %" PRIu64 " with %d\n", seed, streak);
        }
    }
}

int main() {
    uint64_t start = 1ULL;         
    uint64_t end   = 1000000000000ULL;

    int threads = 256;
    int blocks  = 256;

    search<<<blocks, threads>>>(start, end);
    hipDeviceSynchronize();

    return 0;
}