// !nvcc -O3 -arch=sm_75 backender.cu -o backender
// Finding back-to-back pearls

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <inttypes.h>

#define MASK_64         0xFFFFFFFFFFFFFFFFULL
#define SILVER_RATIO_64 0x6a09e667f3bcc909ULL
#define SUBTRACT_CONST  0x61C8864680B583EBULL
#define ENDERMAN_MD5_0  0xa6dd84b6981b3414ULL
#define ENDERMAN_MD5_1  0x4bec48c8de2830b3ULL
#define STAFFORD_MIX_1  0xbf58476d1ce4e5b9ULL
#define STAFFORD_MIX_2  0x94d049bb133111ebULL
#define MIN_SEQUENCE    39

struct Xoroshiro128Plus {
    uint64_t l, h;

    __device__ __forceinline__ Xoroshiro128Plus(uint64_t seed) {
        uint64_t unmixed_lo = seed ^ SILVER_RATIO_64;
        uint64_t unmixed_hi = (unmixed_lo - SUBTRACT_CONST) & MASK_64;

        l = mix_stafford(unmixed_lo ^ ENDERMAN_MD5_0);
        h = mix_stafford(unmixed_hi ^ ENDERMAN_MD5_1);
    }

    __device__ __forceinline__ uint64_t mix_stafford(uint64_t x) {
        x = (x ^ (x >> 30)) * STAFFORD_MIX_1;
        x = (x ^ (x >> 27)) * STAFFORD_MIX_2;
        x ^= x >> 31;
        return x & MASK_64;
    }

    __device__ __forceinline__ uint64_t rotl(uint64_t x, int k) {
        return ((x << k) | (x >> (64 - k))) & MASK_64;
    }

    __device__ __forceinline__ uint64_t next_long() {
        uint64_t s0 = l;
        uint64_t s1 = h;
        uint64_t result = rotl((s0 + s1) & MASK_64, 17) + s0;
        s1 ^= s0;
        l = rotl(s0, 49) ^ s1 ^ ((s1 << 21) & MASK_64);
        h = rotl(s1, 28);
        return result & MASK_64;
    }

    __device__ __forceinline__ uint32_t next_int32() {
        return next_long() & 0xFFFFFFFFULL;
    }

    __device__ __forceinline__ bool next_ender_pearl() {
        return ((next_int32() >> 31) % 2) == 1;
    }
};

__global__ void search(uint64_t start, uint64_t end) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    for (uint64_t seed = start + idx; seed < end; seed += stride) {
        Xoroshiro128Plus rng(seed);

        int streak = 0;
        while (rng.next_ender_pearl()) {
            streak++;
        }

        if (streak >= MIN_SEQUENCE) {
            printf("Seed %" PRIu64 " with %d\n", seed, streak);
        }
    }
}

int main() {
    uint64_t start = 0ULL;
    uint64_t end   = 1000000000000ULL;

    int threads = 256;
    int blocks  = 256;

    search<<<blocks, threads>>>(start, end);
    hipDeviceSynchronize();

    return 0;
}